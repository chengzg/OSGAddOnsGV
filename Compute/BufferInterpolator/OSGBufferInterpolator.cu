#include "hip/hip_runtime.h"
/*---------------------------------------------------------------------------*\
 *                                OpenSG                                     *
 *                                                                           *
 *                                                                           *
 *                 Copyright (C) 2003 by the OpenSG Forum                    *
 *                                                                           *
 *                            www.opensg.org                                 *
 *                                                                           *
 *   contact: dirk@opensg.org, gerrit.voss@vossg.org, jbehr@zgdv.de          *
 *                                                                           *
\*---------------------------------------------------------------------------*/
/*---------------------------------------------------------------------------*\
 *                                License                                    *
 *                                                                           *
 * This library is free software; you can redistribute it and/or modify it   *
 * under the terms of the GNU Library General Public License as published    *
 * by the Free Software Foundation, version 2.                               *
 *                                                                           *
 * This library is distributed in the hope that it will be useful, but       *
 * WITHOUT ANY WARRANTY; without even the implied warranty of                *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU         *
 * Library General Public License for more details.                          *
 *                                                                           *
 * You should have received a copy of the GNU Library General Public         *
 * License along with this library; if not, write to the Free Software       *
 * Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.                 *
 *                                                                           *
\*---------------------------------------------------------------------------*/
/*---------------------------------------------------------------------------*\
 *                                Changes                                    *
 *                                                                           *
 *                                                                           *
 *                                                                           *
 *                                                                           *
 *                                                                           *
 *                                                                           *
\*---------------------------------------------------------------------------*/

#include "OSGBufferInterpolator.cuh"

OSGCUDA_BEGIN_NAMESPACE

__global__ 
void linearBufferInterpolate(const float *pInLower,
                             const float *pInUpper,
                             const float  t,
                                   int    numElements,
                                   float *pResult    )
{
    const int        tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    float tmp = 0.f;

    for(int pos = tid; pos < numElements; pos += numThreads)
    {
        tmp = pInUpper[pos] - pInLower[pos];

        pResult[pos] = pInLower[pos] + (t * tmp);
    }
}


void lerpFloat(const float *pInLower,
               const float *pInUpper,
               const float  t,
                     int    numElements,
                     float *pResult)

{
    linearBufferInterpolate<<<64, 256>>>(pInLower,
                                         pInUpper,
                                         t,
                                         numElements,
                                         pResult);
}

OSGCUDA_END_NAMESPACE
