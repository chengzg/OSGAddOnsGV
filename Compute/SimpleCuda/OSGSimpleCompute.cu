#include "hip/hip_runtime.h"
/*---------------------------------------------------------------------------*\
 *                                OpenSG                                     *
 *                                                                           *
 *                                                                           *
 *                 Copyright (C) 2003 by the OpenSG Forum                    *
 *                                                                           *
 *                            www.opensg.org                                 *
 *                                                                           *
 *   contact: dirk@opensg.org, gerrit.voss@vossg.org, jbehr@zgdv.de          *
 *                                                                           *
\*---------------------------------------------------------------------------*/
/*---------------------------------------------------------------------------*\
 *                                License                                    *
 *                                                                           *
 * This library is free software; you can redistribute it and/or modify it   *
 * under the terms of the GNU Library General Public License as published    *
 * by the Free Software Foundation, version 2.                               *
 *                                                                           *
 * This library is distributed in the hope that it will be useful, but       *
 * WITHOUT ANY WARRANTY; without even the implied warranty of                *
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU         *
 * Library General Public License for more details.                          *
 *                                                                           *
 * You should have received a copy of the GNU Library General Public         *
 * License along with this library; if not, write to the Free Software       *
 * Foundation, Inc., 675 Mass Ave, Cambridge, MA 02139, USA.                 *
 *                                                                           *
\*---------------------------------------------------------------------------*/
/*---------------------------------------------------------------------------*\
 *                                Changes                                    *
 *                                                                           *
 *                                                                           *
 *                                                                           *
 *                                                                           *
 *                                                                           *
 *                                                                           *
\*---------------------------------------------------------------------------*/

#include "OSGSimpleCompute.cuh"

OSGCUDA_BEGIN_NAMESPACE

#if 0

__global__ 
void linearBufferInterpolate(const float *pInLower,
                             const float *pInUpper,
                             const float  t,
                                   int    numElements,
                                   float *pResult    )
{
    const int        tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    float tmp = 0.f;

    for(int pos = tid; pos < numElements; pos += numThreads)
    {
        tmp = pInUpper[pos] - pInLower[pos];

        pResult[pos] = pInLower[pos] + (t * tmp);
    }
}


void lerpFloat(const float *pInLower,
               const float *pInUpper,
               const float  t,
                     int    numElements,
                     float *pResult)

{
    linearBufferInterpolate<<<64, 256>>>(pInLower,
                                         pInUpper,
                                         t,
                                         numElements,
                                         pResult);
}
#endif

struct CudaFloat3 
{
    float x, y, z;
};

__global__ 
void simpleComputeUpdateGeoCuda(float         rScale,
                                float         rIntensity,
                                unsigned int  uiNumPnts,
                                CudaFloat3   *pMappedPoints,
                                CudaFloat3   *pMappedNormals)
{
    const int        tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    for(int pos = tid; pos < uiNumPnts; pos += numThreads)
    {
        float rDist = sqrt((pMappedPoints[pos].x * pMappedPoints[pos].x) + 
                           (pMappedPoints[pos].y * pMappedPoints[pos].y));

        float rBump = rIntensity * cos(rDist * rScale);

        pMappedPoints[pos].z = rBump;


        rBump = rIntensity * (rScale/(rDist)) * -2 * -sin(rDist * rScale);

        float tmp[3];

        tmp[0] = rBump * pMappedPoints[pos].x;
        tmp[1] = rBump * pMappedPoints[pos].y;
        tmp[2] = 1.f;

        rDist = 1.f / sqrt(tmp[0] * tmp[0] + tmp[1] * tmp[1] + tmp[2] * tmp[2]);

        pMappedNormals[pos].x = tmp[0] * rDist;
        pMappedNormals[pos].y = tmp[1] * rDist;
        pMappedNormals[pos].z = tmp[2] * rDist;
    }
}

void simpleComputeUpdateGeo(float         rScale,
                            float         rIntensity,
                            unsigned int  uiNumPnts,
                            void         *pMappedPoints,
                            void         *pMappedNormals)
{
    CudaFloat3 *pPnts  = static_cast<CudaFloat3 *>(pMappedPoints );
    CudaFloat3 *pNorms = static_cast<CudaFloat3 *>(pMappedNormals);

    simpleComputeUpdateGeoCuda<<<64, 256>>>(rScale,
                                            rIntensity,
                                            uiNumPnts,
                                            pPnts,
                                            pNorms);
}

OSGCUDA_END_NAMESPACE
